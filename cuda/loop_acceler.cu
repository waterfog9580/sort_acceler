
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{
  
  printf("This is iteration number %d\n", threadIdx.x);
  
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */

  int N = 10;
  loop<<<1,N>>>();
  hipDeviceSynchronize();
}

/*
kernel은 해당 반복문에서 딱 한 번의 반복 작업만 하도록 설계되어야 한다.
kernel이 다른 kernel에 대해서 알지 못하기 때문에, execution configuration이 해당 반복문에서 반복되는 작업의 수에 맞춰 선언되어야 한다.
*/