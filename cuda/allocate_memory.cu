
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__ //Define this function is computed by gpu(eg, __global__ void function())
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 100;
  int *a;

  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);  //Array allocate for GPU
  init(a, N);                   //Array allocate for CPU

  size_t threads_per_block = 10;
  size_t number_of_blocks = 10;


  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N); //GPU compute
  hipDeviceSynchronize(); //synchronize GPU with CPU 

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}

/*
kernel: GPU function을 부르는 용어. kernel은 execution configuration에 따라 실행된다.
thread: GPU 작업의 기본 단위. 여러 thread가 병렬적으로 작동한다.
block:  thread의 모임을 block이라 한다.
grid:   주어진 kernel의 execution configuration에서 block들의 모임, 그러니까 전체를 grid라 부른다.
*/